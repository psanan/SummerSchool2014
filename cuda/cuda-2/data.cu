#include "data.h"

#include <stdio.h>

struct discretization_t cpu::options;

hipDeviceProp_t cpu::props;

// fields that hold the solution
__constant__ double *gpu::x_old = NULL; // 2d
__constant__ double *gpu::bndN = NULL, *gpu::bndE = NULL;
__constant__ double *gpu::bndS = NULL, *gpu::bndW = NULL;

__constant__ struct discretization_t gpu::options;

