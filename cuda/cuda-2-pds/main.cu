#include "hip/hip_runtime.h"
// ******************************************
// implicit time stepping implementation of 2D diffusion problem
// Ben Cumming, CSCS
// C version by Gilles Fourestey, CSCS
// *****************************************

// A small benchmark app that solves the 2D fisher equation using second-order
// finite differences.

// Syntax: ./main nx ny nt t

#include <assert.h>
#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "check.h"
#include "data.h"
#include "linalg.h"
#include "operators.h"
#include "stats.h"

// ==============================================================================

// read command line arguments
static void readcmdline(struct discretization_t* options, int argc, char* argv[])
{
	if (argc != 5)
	{
		printf("Usage: main nx ny nt t\n");
		printf("  nx  number of gridpoints in x-direction\n");
		printf("  ny  number of gridpoints in y-direction\n");
		printf("  nt  number of timesteps\n");
		printf("  t   total time\n");
		exit(1);
	}

	// read nx
	options->nx = atoi(argv[1]);
	if (options->nx < 1)
	{
		fprintf(stderr, "nx must be positive integer\n");
		exit(-1);
	}

	// read ny
	options->ny = atoi(argv[2]);
	if (options->ny < 1)
	{
		fprintf(stderr, "ny must be positive integer\n");
		exit(-1);
	}

	// read nt
	options->nt = atoi(argv[3]);
	if (options->nt < 1)
	{
		fprintf(stderr, "nt must be positive integer\n");
		exit(-1);
	}
	
	// read total time
	double t = atof(argv[4]);
	if (t < 0)
	{
		fprintf(stderr, "t must be positive real value\n");
		exit(-1);
	}

	// store the parameters
	options->N = options->nx * options->ny;

	// compute timestep size
	options->dt = t / options->nt;
	
	// compute the distance between grid points
	// assume that x dimension has length 1.0
	options->dx = 1./(options->nx - 1);
	
	// set alpha, assume diffusion coefficient D is 1
	options->alpha = (options->dx*options->dx) / (1.*options->dt);
}

// ==============================================================================

int main(int argc, char* argv[])
{
	// read command line arguments
	readcmdline(&cpu::options, argc, argv);
    CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(
    	gpu::options), &cpu::options, sizeof(struct discretization_t)));

	int nx = cpu::options.nx;
	int ny = cpu::options.ny;
	int N  = cpu::options.N;
	int nt = cpu::options.nt;

	printf("========================================================================\n");
	printf("					  Welcome to mini-stencil!\n");
	printf("mesh :: %d * %d, dx = %f\n", nx, ny, cpu::options.dx);
	printf("time :: %d, time steps from 0 .. %f\n", nt, nt * cpu::options.dt);
	printf("========================================================================\n");

	// allocate global fields
	double* cpu_x_new  = (double*)malloc(sizeof(double) * nx * ny);
	{
		using namespace cpu;

		// set the initial condition
		// a circle of concentration 0.1 centred at (xdim/4, ydim/4) with radius
		// no larger than 1/8 of both xdim and ydim
		memset(cpu_x_new, 0, sizeof(double) * nx * ny);
		double xc = 1.0 / 4.0;
		double yc = (ny - 1) * options.dx / 4;
		double radius = fmin(xc, yc) / 2.0;
		for (int j = 0; j < ny; j++)
		{
			double y = (j - 1) * options.dx;
			for (int i = 0; i < nx; i++)
			{
				double x = (i - 1) * options.dx;
				if ((x - xc) * (x - xc) + (y - yc) * (y - yc) < radius * radius)
					cpu_x_new[i + j * nx] = 0.1;
			}
		}
	}

	CUDA_ERR_CHECK(hipGetDeviceProperties(&cpu::props, 0));
	
	// Calibrating kernels compute grids for the given problem dimensions.
	{
		determine_optimal_grid_block_config(diffusion_interior_grid_points, 1, nx, ny);
		determine_optimal_grid_block_config(diffusion_east_west_boundary_points, 1, 1, ny - 2);
		determine_optimal_grid_block_config(diffusion_north_south_boundary_points, 1, nx - 2, 1);
		determine_optimal_grid_block_configs_reduction(ss_sum, 1, N);
		determine_optimal_grid_block_configs_reduction(ss_dot, 1, N);
		determine_optimal_grid_block_configs_reduction(ss_norm2, 1, N);
		determine_optimal_grid_block_config(ss_fill, 1, N, 1);
		determine_optimal_grid_block_config(ss_axpy, 1, N, 1);
		determine_optimal_grid_block_config(ss_add_scaled_diff, 1, N, 1);
		determine_optimal_grid_block_config(ss_scaled_diff, 1, N, 1);
		determine_optimal_grid_block_config(ss_scale, 1, N, 1);
		determine_optimal_grid_block_config(ss_lcomb, 1, N, 1);
		determine_optimal_grid_block_config(ss_copy, 1, N, 1);
	}

	double *x_old_h, *bndN_h, *bndS_h, *bndE_h, *bndW_h;
	CUDA_ERR_CHECK(hipMalloc(&x_old_h,  sizeof(double) * nx * ny));
	CUDA_ERR_CHECK(hipMalloc(&bndN_h,   sizeof(double) * nx));
	CUDA_ERR_CHECK(hipMalloc(&bndS_h,   sizeof(double) * nx));
	CUDA_ERR_CHECK(hipMalloc(&bndE_h,   sizeof(double) * ny));
	CUDA_ERR_CHECK(hipMalloc(&bndW_h,   sizeof(double) * ny));
	
	using namespace gpu;
	
	CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(x_old), &x_old_h, sizeof(double*)));
	CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bndN), &bndN_h, sizeof(double*)));
	CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bndS), &bndS_h, sizeof(double*)));
	CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bndE), &bndE_h, sizeof(double*)));
	CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bndW), &bndW_h, sizeof(double*)));

	double *b;
	CUDA_ERR_CHECK(hipMalloc(&b,	  sizeof(double) * N));
	double *deltax;
	CUDA_ERR_CHECK(hipMalloc(&deltax, sizeof(double) * N));
	
	// setting up shmem-cached flops counters
	flops_diff = 0, flops_blas1 = 0;
	iters_cg = 0; iters_newton = 0;
	
	// set dirichlet boundary conditions to 0 all around
	ss_fill(get_value(x_old),  0, N);
	ss_fill(get_value(bndN),   0, nx);
	ss_fill(get_value(bndS),   0, nx);
	ss_fill(get_value(bndE),   0, ny);
	ss_fill(get_value(bndW),   0, ny);
	ss_fill(deltax, 0, N);

	// start timer
	double timespent = -omp_get_wtime();

	// copy initial solution to GPU
	double* gpu_x_new;
	CUDA_ERR_CHECK(hipMalloc(&gpu_x_new, sizeof(double) * nx * ny));
	if (gpu_x_new == NULL) assert(false);
	CUDA_ERR_CHECK(hipMemcpy(gpu_x_new, cpu_x_new, sizeof(double) * nx * ny, hipMemcpyHostToDevice));
	double* x_new = gpu_x_new;

	// main timeloop
	double tolerance = 1.e-6;
	int timestep;
	for (timestep = 1; timestep <= nt; timestep++)
	{
		// set x_new and x_old to be the solution
		ss_copy(get_value(x_old), x_new, N);

		double residual;
		int	converged = 0;
		int	it = 1;
		for ( ; it <= 50; it++)
		{
			// compute residual : requires both x_new and x_old
			diffusion(x_new, b);
			residual = ss_norm2(b, N);

			// check for convergence
			if (residual < tolerance)
			{
				converged = 1;
				break;
			}

			// solve linear system to get -deltax
			bool cg_converged = ss_cg(N, deltax, b, 200, tolerance);

			// check that the CG solver converged
			if (!cg_converged) break;

			// update solution
			ss_axpy(x_new, -1.0, deltax, N);

			// print control sum of x_new
			if (timestep % 50 == 0)
			{
				double sum = ss_sum(x_new, N);
				printf("sum = %f\n", sum);
			}
		}
		iters_newton += it;

		// output some statistics
		if (converged && verbose_output)
			printf("step %d required %d iterations for residual %E\n", timestep, it, residual);
		if (!converged)
		{
			printf("step %d ERROR : nonlinear iterations failed to converge\n", timestep);
			break;
		}
	}
	
	CUDA_ERR_CHECK(hipMemcpy(cpu_x_new, gpu_x_new, sizeof(double) * nx * ny, hipMemcpyDeviceToHost));

	// get times
	timespent += omp_get_wtime();
	unsigned long long flops_total = flops_diff + flops_blas1;

	hipFree(get_value(x_old));
	hipFree(get_value(bndN));
	hipFree(get_value(bndS));
	hipFree(get_value(bndE));
	hipFree(get_value(bndW));
	hipFree(b);
	hipFree(deltax);

	using namespace cpu;

	////////////////////////////////////////////////////////////////////
	// write final solution to BOV file for visualization
	////////////////////////////////////////////////////////////////////

	// binary data
	{
		FILE* output = fopen("output.bin", "w");
		fwrite(cpu_x_new, sizeof(double), nx * ny, output);
		fclose(output);
	}

	// metadata
	{
		FILE* output = fopen("output.bov", "wb");
		fprintf(output, "TIME: 0.0\n");
		fprintf(output, "DATA_FILE: output.bin\n");
		fprintf(output, "DATA_SIZE: %d, %d, 1\n", nx, ny);
		fprintf(output, "DATA_FORMAT: DOUBLE\n");
		fprintf(output, "VARIABLE: phi\n");
		fprintf(output, "DATA_ENDIAN: LITTLE\n");
		fprintf(output, "CENTERING: nodal\n");
		//fprintf(output, "BYTE_OFFSET: 4\n");
		fprintf(output, "BRICK_SIZE: 1.0 %f 1.0\n", (ny - 1) * cpu::options.dx);
		fclose(output);
	}

	// print table sumarizing results
	printf("--------------------------------------------------------------------------------\n");
	printf("simulation took %f seconds (%f GFLOP/s)\n", timespent, flops_total / 1e9 / timespent);
	printf("%u conjugate gradient iterations\n", iters_cg);
	printf("%u newton iterations\n", iters_newton);
	printf("--------------------------------------------------------------------------------\n");

	// deallocate global fields
	CUDA_ERR_CHECK(hipFree(gpu_x_new));
	free(cpu_x_new);

	printf("Goodbye!\n");

	return 0;
}

