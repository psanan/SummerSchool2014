#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_ERR_CHECK(x) \
	do { hipError_t err = x; if (err != hipSuccess) { \
		fprintf (stderr, "Error \"%s\" at %s:%d \n", \
		hipGetErrorString(err), \
		__FILE__, __LINE__); exit(-1); \
	}} while (0);

#define roundup(n, width) (((n) + (width) - 1) & ~unsigned((width) - 1))
#define rounddown(n, width) (n / width * width)

__global__ void gpu_kernel(int ni, int nj, int nk, int* data, int i_inc, int j_inc, int k_inc)
{
	for (int k = blockIdx.z * blockDim.z + threadIdx.z; k < nk; k += k_inc)
		for (int j = blockIdx.y * blockDim.y + threadIdx.y; j < nj; j += j_inc)
			for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < ni; i += i_inc)
			{
				int idx = i + ni * j + nj * ni * k;
				assert(data[idx] == 0);
				data[idx] = idx;
			}
}

int main(int argc, char* argv[])
{
	if (argc != 4)
	{
		printf("Usage: %s <ni> <nj> <nk>\n", argv[0]);
		return 0;
	}

	int ni = atoi(argv[1]);
	int nj = atoi(argv[2]);
	int nk = atoi(argv[3]);

	int* data = (int*)malloc(ni * nj * nk * sizeof(int));
	for (int k = 0; k < nk; k++)
		for (int j = 0; j < nj; j++)
			for (int i = 0; i < ni; i++)
			{
				int idx = i + ni * j + nj * ni * k;
				data[idx] = idx;
			}

	int* gpu_data;
	CUDA_ERR_CHECK( hipMalloc(&gpu_data, ni * nj * nk * sizeof(int)) );
	CUDA_ERR_CHECK( hipMemset(gpu_data, 0, ni * nj * nk * sizeof(int)) );

	struct hipDeviceProp_t props;
	CUDA_ERR_CHECK( hipGetDeviceProperties(&props, 0) );
	dim3 max_grid;
	max_grid.x = props.maxGridSize[0];
	max_grid.y = props.maxGridSize[1];
	max_grid.z = props.maxGridSize[2];
	
	gpu_kernel<<<dim3(min(max(1, roundup(ni, 16) / 16), rounddown(max_grid.x, 16)),
	                  min(max(1, roundup(nj,  8) /  8), rounddown(max_grid.y,  8)),
	                  min(max(1, roundup(nk,  8) /  8), rounddown(max_grid.z,  8))),
	             dim3(16, 8, 8)>>>(ni, nj, nk, gpu_data,
	                  min(max(1, roundup(ni, 16) / 16), rounddown(max_grid.x, 16)) * 16,
	                  min(max(1, roundup(nj,  8) /  8), rounddown(max_grid.y,  8)) * 8,
	                  min(max(1, roundup(nk,  8) /  8), rounddown(max_grid.z,  8)) * 8);

	CUDA_ERR_CHECK( hipGetLastError() );
	
	int* host_data = (int*)malloc(ni * nj * nk * sizeof(int));
	CUDA_ERR_CHECK( hipMemcpy(host_data, gpu_data, ni * nj * nk * sizeof(int),
		hipMemcpyDeviceToHost) );
	CUDA_ERR_CHECK( hipFree(gpu_data) );
	
	for (int k = 0; k < nk; k++)
		for (int j = 0; j < nj; j++)
			for (int i = 0; i < ni; i++)
			{
				int idx = i + ni * j + nj * ni * k;
				if (data[idx] != host_data[idx])
				{
					fprintf(stderr, "Values mismatch at (i, j, k) = (%d, %d, %d): %d != %d\n",
						i, j, k, data[idx], host_data[idx]);
					exit(1);
				}
			}

	printf("Done!\n");

	free(host_data);
	free(data);

	return 0;
}
